#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023 Rick Kern <kernrj@gmail.com>
 *
 * This program is free software: you can redistribute it and/or modify it under the terms of the GNU Affero General
 * Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any
 * later version.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied
 * warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Affero General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Affero General Public License along with this program.  If not, see
 * <https://www.gnu.org/licenses/>.
 */

#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>

#include "cuComplexOperatorOverloads.cuh"
#include "gsdr/cuda_util.h"
#include "gsdr/fir.h"

using namespace std;

template <class IN_T, class OUT_T, class TAP_T>
__global__ void k_Fir(
    const IN_T* input,
    const TAP_T* tapsReversed,
    uint32_t numTaps,
    OUT_T* output,
    uint32_t numOutputs) {
  uint32_t outputIndex = blockDim.x * blockIdx.x + threadIdx.x;
  uint32_t inputIndex = outputIndex;

  if (outputIndex >= numOutputs) {
    return;
  }

  OUT_T& outputSample = output[outputIndex];
  const IN_T* inputSample = input + inputIndex;

  outputSample = zero<OUT_T>();
  for (uint32_t i = 0; i < numTaps; i++, inputSample++) {
    outputSample += *inputSample * tapsReversed[i];
  }
}

template <class IN_T, class OUT_T, class TAP_T>
__global__ void k_FirDecimate(
    const IN_T* __restrict__ input,
    const TAP_T* __restrict__ tapsReversed,
    uint32_t numTaps,
    uint32_t decimation,
    OUT_T* __restrict__ output,
    uint32_t numOutputs) {
  uint32_t outputIndex = blockDim.x * blockIdx.x + threadIdx.x;
  uint32_t inputIndex = decimation * outputIndex;

  if (outputIndex >= numOutputs) {
    return;
  }

  OUT_T& outputSample = output[outputIndex];
  const IN_T* inputSample = input + inputIndex;
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 200
  // printf("outputIndex [%u] inputIndex [%u]\n", outputIndex, inputIndex);
#endif

  outputSample = zero<OUT_T>();
  for (uint32_t i = 0; i < numTaps; i++, inputSample++) {
    outputSample += *inputSample * tapsReversed[i];
  }
}

GSDR_C_LINKAGE hipError_t gsdrFirFC(
    size_t decimation,
    const float* taps,
    size_t tapCount,
    const hipComplex* input,
    hipComplex* output,
    size_t numOutputs,
    int32_t cudaDevice,
    hipStream_t cudaStream) GSDR_NO_EXCEPT {
  const size_t numElements = numOutputs;
  SIMPLE_CUDA_FNC_START("FIR FC")

  if (decimation == 1) {
    CHECK_CUDA_RET("Before k_Fir() FC");
    k_Fir<<<blocks, threads, 0, cudaStream>>>(input, taps, tapCount, output, numOutputs);
    CHECK_CUDA_RET("After k_Fir() FC");
  } else {
    CHECK_CUDA_RET("Before k_FirDecimate() FC");
    k_FirDecimate<<<blocks, threads, 0, cudaStream>>>(input, taps, tapCount, decimation, output, numOutputs);
    CHECK_CUDA_RET("After k_FirDecimate() FC");
  }

  SIMPLE_CUDA_FNC_END("FIR FC")
}

GSDR_C_LINKAGE hipError_t gsdrFirFF(
    size_t decimation,
    const float* taps,
    size_t tapCount,
    const float* input,
    float* output,
    size_t numOutputs,
    int32_t cudaDevice,
    hipStream_t cudaStream) GSDR_NO_EXCEPT {
  const size_t numElements = numOutputs;
  SIMPLE_CUDA_FNC_START("FIR FF")

  if (decimation == 1) {
    CHECK_CUDA_RET("Before k_Fir() FF");
    k_Fir<<<blocks, threads, 0, cudaStream>>>(input, taps, tapCount, output, numOutputs);
    CHECK_CUDA_RET("After k_Fir() FF");
  } else {
    CHECK_CUDA_RET("Before k_FirDecimate() FF");
    k_FirDecimate<<<blocks, threads, 0, cudaStream>>>(input, taps, tapCount, decimation, output, numOutputs);
    CHECK_CUDA_RET("After k_FirDecimate() FF");
  }

  SIMPLE_CUDA_FNC_END("FIR FF")
}

GSDR_C_LINKAGE hipError_t gsdrFirCC(
    size_t decimation,
    const hipComplex* taps,
    size_t tapCount,
    const hipComplex* input,
    hipComplex* output,
    size_t numOutputs,
    int32_t cudaDevice,
    hipStream_t cudaStream) GSDR_NO_EXCEPT {
  const size_t numElements = numOutputs;
  SIMPLE_CUDA_FNC_START("FIR CC")

  if (decimation == 1) {
    CHECK_CUDA_RET("Before k_Fir() CC");
    k_Fir<<<blocks, threads, 0, cudaStream>>>(input, taps, tapCount, output, numOutputs);
    CHECK_CUDA_RET("After k_Fir() CC");
  } else {
    CHECK_CUDA_RET("Before k_FirDecimate() CC");
    k_FirDecimate<<<blocks, threads, 0, cudaStream>>>(input, taps, tapCount, decimation, output, numOutputs);
    CHECK_CUDA_RET("After k_FirDecimate() CC");
  }

  SIMPLE_CUDA_FNC_END("FIR CC")
}

GSDR_C_LINKAGE hipError_t gsdrFirCF(
    size_t decimation,
    const hipComplex* taps,
    size_t tapCount,
    const float* input,
    hipComplex* output,
    size_t numOutputs,
    int32_t cudaDevice,
    hipStream_t cudaStream) GSDR_NO_EXCEPT {
  const size_t numElements = numOutputs;
  SIMPLE_CUDA_FNC_START("FIR CF")

  if (decimation == 1) {
    CHECK_CUDA_RET("Before k_Fir() CF");
    k_Fir<<<blocks, threads, 0, cudaStream>>>(input, taps, tapCount, output, numOutputs);
    CHECK_CUDA_RET("After k_Fir() CF");
  } else {
    CHECK_CUDA_RET("Before k_FirDecimate() CF");
    k_FirDecimate<<<blocks, threads, 0, cudaStream>>>(input, taps, tapCount, decimation, output, numOutputs);
    CHECK_CUDA_RET("After k_FirDecimate() CF");
  }

  SIMPLE_CUDA_FNC_END("FIR CF")
}
