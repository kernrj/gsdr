#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023 Rick Kern <kernrj@gmail.com>
 *
 * This program is free software: you can redistribute it and/or modify it under the terms of the GNU Affero General
 * Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any
 * later version.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied
 * warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Affero General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Affero General Public License along with this program.  If not, see
 * <https://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>

#include "cuComplexOperatorOverloads.cuh"
#include "gsdr/quad_demod.h"
#include "gsdr/util.h"

__global__ static void k_quadFmDemod(const hipComplex* input, float* output, float gain, uint32_t numOutputElements) {
  uint32_t index = blockDim.x * blockIdx.x + threadIdx.x;

  if (index >= numOutputElements) {
    return;
  }

  const hipComplex m = input[index + 1] * hipConjf(input[index]);
  const float outputValue = gain * atan2f(m.y, m.x);
  output[index] = outputValue;

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 200
  // printf("index [%u] outputValue [%f]\n", index, outputValue);
#endif
}

__global__ static void k_quadAmDemod(const hipComplex* input, float* output, uint32_t numOutputElements) {
  uint32_t index = blockDim.x * blockIdx.x + threadIdx.x;

  if (index >= numOutputElements) {
    return;
  }

  hipComplex val = input[index];
  float magnitude = hypotf(val.x, val.y);
  float outputValue = scalbnf(__saturatef(magnitude), 1) - 1.0f;
  output[index] = outputValue;

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 200
  // printf("index [%u] magnitude [%f] amplitude [%f]\n", index, magnitude, outputValue);
#endif
}

GSDR_C_LINKAGE hipError_t gsdrQuadFmDemod(
    const hipComplex* input,
    float* output,
    float gain,
    size_t numElements,
    int32_t cudaDevice,
    hipStream_t cudaStream) GSDR_NO_EXCEPT {
  SIMPLE_CUDA_FNC_START("k_quadFmDemod()")
  k_quadFmDemod<<<blocks, threads, 0, cudaStream>>>(input, output, gain, numElements);
  SIMPLE_CUDA_FNC_END("k_quadFmDemod()")
}

GSDR_C_LINKAGE hipError_t
gsdrQuadAmDemod(const hipComplex* input, float* output, size_t numElements, int32_t cudaDevice, hipStream_t cudaStream)
    GSDR_NO_EXCEPT {
  SIMPLE_CUDA_FNC_START("k_quadAmDemod()")
  k_quadAmDemod<<<blocks, threads, 0, cudaStream>>>(input, output, numElements);
  SIMPLE_CUDA_FNC_END("k_quadAmDemod()")
}
