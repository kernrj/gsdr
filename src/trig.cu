#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023 Rick Kern <kernrj@gmail.com>
 *
 * This program is free software: you can redistribute it and/or modify it under the terms of the GNU Affero General
 * Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any
 * later version.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied
 * warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Affero General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Affero General Public License along with this program.  If not, see
 * <https://www.gnu.org/licenses/>.
 */
#include "cuComplexOperatorOverloads.cuh"
#include "gsdr/trig.h"

__global__ void k_ComplexCosine(float indexToRadiansMultiplier, float phi, hipComplex* values, size_t numElements) {
  const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
  if (x > numElements) {
    return;
  }

  const float theta = phi + __uint2float_rn(x) * indexToRadiansMultiplier;

  hipComplex result;
  sincosf(theta, &result.x, &result.y);

  values[x] = result;
}

C_LINKAGE hipError_t gsdrCosineC(
    float phiBegin,
    float phiEnd,
    hipComplex* output,
    size_t numElements,
    int32_t cudaDevice,
    hipStream_t cudaStream) {
  SIMPLE_CUDA_FNC_START("k_ComplexCosine()");

  const auto indexToRadiansMultiplier = static_cast<float>((phiEnd - phiBegin) / static_cast<double>(numElements));
  k_ComplexCosine<<<blocks, threads, 0, cudaStream>>>(indexToRadiansMultiplier, phiBegin, output, numElements);

  SIMPLE_CUDA_FNC_END("k_ComplexCosine()");
}
